#include "hip/hip_runtime.h"


#ifndef _BACKPROP_CUDA_KERNEL_H_
#define _BACKPROP_CUDA_KERNEL_H_

#include <stdio.h>
#include "backprop.h"
#include "math.h"
#include "hip/hip_runtime.h"

__global__ void
bpnn_layerforward_CUDA(float *input_cuda,
	                   float *output_hidden_cuda,
					   float *input_hidden_cuda,
					   float *hidden_partial_sum,
					   int in,
					   int hid) 
{
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int index = ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  

   int index_in = HEIGHT * by + ty + 1;
   
  //  __shared__ float input_node[HEIGHT];
   __shared__ float weight_matrix[HEIGHT][WIDTH];

   weight_matrix[ty][tx] = input_hidden_cuda[index] * input_cuda[index_in];

   __syncthreads();   

  if((ty & 1) ==0){
    weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 1][tx];
    if((ty & 3) ==0){
      weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 2][tx];
      if((ty & 7) ==0){
        weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 4][tx];
        if((ty & 15) ==0){
          weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 8][tx];
        }
      }
    }
  }
    


   input_hidden_cuda[index] = weight_matrix[ty][tx];

   if (tx == 0) {
	   hidden_partial_sum[by * hid + ty] = weight_matrix[tx][ty];
   }

}



__global__ void bpnn_adjust_weights_cuda(float * delta,   
										 int hid,         
										 float * ly,      
										 int in,          
										 float * w,       
										 float * oldw)  									
{
  
  
   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;
	
   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
   oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));


   __syncthreads();

   if (ty == 0 && by ==0){
   w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   }


}
#endif 
