#include "hip/hip_runtime.h"


#ifndef _BACKPROP_CUDA_KERNEL_H_
#define _BACKPROP_CUDA_KERNEL_H_

#include <stdio.h>
#include "backprop.h"
#include "math.h"
#include "hip/hip_runtime.h"

__global__ void
bpnn_layerforward_CUDA(float *input_cuda,
	                   float *output_hidden_cuda,
					   float *input_hidden_cuda,
					   float *hidden_partial_sum,
					   int in,
					   int hid) 
{
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int index = ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  

   int index_in = HEIGHT * by + ty + 1;
   
  //  __shared__ float input_node[HEIGHT];
  //  __shared__ float weight_matrix[HEIGHT][WIDTH];
  __shared__ float weight_matrix[WIDTH];
  //  weight_matrix[ty][tx] = input_hidden_cuda[index] * input_cuda[index_in];

  float r1 = input_hidden_cuda[index] * input_cuda[index_in];
  float r2, r3, r4, r5, r6, r7, r8, r9, rfinal;
  //  __syncthreads();   
   
  //  for ( unsigned int i = 2 ; i <= HEIGHT ; i *= 2){
  //    unsigned int power_two = i - 1;

  //    if( (ty & power_two) == 0 ) {
  //      weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];
  //    }

  //    __syncthreads();
  //  }


  //  float r1 = weight_matrix[ty][tx];
  //  float r2 = weight_matrix[ty+1][tx];

  

  if((ty & 1) ==0){
    r2 = __shfl_down_sync(0xffffffff, r1, 1);
    r3 = r1+r2;
    // weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 1][tx];
    if((ty & 3) ==0){
      r4 = __shfl_down_sync(0xffffffff, r3, 3);
      // weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 2][tx];
      r5 = r3+r4;
      if((ty & 7) ==0){
        r6 = __shfl_down_sync(0xffffffff, r5, 7);
        r7 = r5+r6;
        // weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 4][tx];
        if((ty & 15) ==0){
          r8 = __shfl_down_sync(0xffffffff, r7, 15);
          
          rfinal = r7+r8;
          // weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + 8][tx];
        }else{
          rfinal = r7;
        }
      }else{
        rfinal = r5;
      }
    }else{
      rfinal = r3;
    }
  }else{
    rfinal = r1;
  }
  input_hidden_cuda[index] = rfinal;
  //  input_hidden_cuda[index] = weight_matrix[ty][tx];
  if(ty==0){
    weight_matrix[tx] = rfinal;
  __syncthreads();
  }
   if (tx == 0) {
	   hidden_partial_sum[by * hid + ty] = weight_matrix[ty];
   }

}



__global__ void bpnn_adjust_weights_cuda(float * delta,   
										 int hid,         
										 float * ly,      
										 int in,          
										 float * w,       
										 float * oldw)  									
{
  
  
   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;
	
   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
   oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));


   __syncthreads();

   if (ty == 0 && by ==0){
   w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   }


}
#endif 
